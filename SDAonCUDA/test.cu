#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include ""
#include "SDAonCUDA.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace GPU
{
	__global__ void gpuSDAmultiDim(uint8_t* in, uint8_t* out, uint32_t frames, uint32_t height, uint32_t width, float radius, uint16_t iradius, int threshold, uint64_t size)
	{
		uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
		uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
		uint32_t z = threadIdx.z + blockIdx.z * blockDim.z;

		if (x > width || y > height || z > frames)
			return;

		for (int16_t k = -iradius; k <= iradius; k++)
			if (0 <= z + k && z + k < frames)
				for (int16_t j = -iradius; j <= iradius; j++)
					if (0 <= y + j && y + j < height)
						for (int16_t i = -iradius; i <= iradius; i++)
							if (i * i + j * j + k * k <= radius * radius && 0 <= x + i && x + i < width)
								if (in[((z + k) * height + y + j) * width + x + i] >= in[(z * height + y) * width + x] + threshold)
									out[(z * height + y) * width + x]++;
	}
	__global__ void gpuSDAsingleDim(uint8_t* in, uint8_t* out, uint32_t frames, uint32_t height, uint32_t width, float radius, uint16_t iradius, int threshold, uint64_t size)
	{
		uint64_t tempid = threadIdx.x + blockIdx.x * blockDim.x;
 		uint32_t x = (tempid) % width;
		tempid /= width;
		uint32_t y = tempid % height;
		tempid /= height;
		uint32_t z = tempid % frames;

		if (tempid / frames > 0)
			return;

		for (int16_t k = -iradius; k <= iradius; k++)
			if (0 <= z + k && z + k < frames)
				for (int16_t j = -iradius; j <= iradius; j++)
					if (0 <= y + j && y + j < height)
						for (int16_t i = -iradius; i <= iradius; i++)
							if (i * i + j * j + k * k <= radius * radius && 0 <= x + i && x + i < width)
								if (in[((z + k) * height + y + j) * width + x + i] >= in[(z * height + y) * width + x] + threshold)
									out[(z * height + y) * width + x]++;
	}

	//template<class InBitDepth, class OutBitDepth>
	//void GpuSDA(InBitDepth* image, OutBitDepth* output, float radius, int threshold)
	void SDA(uint8_t* input, uint8_t* output, float radius, int threshold, uint32_t frames, uint32_t height, uint32_t width)
	{
		//hipDeviceProp_t prop;
		//hipGetDeviceProperties(&prop, 0);
		//std::cout << "\ngrid:" << prop.maxGridSize[0] << "\n" << prop.maxGridSize[1] << "\n" << prop.maxGridSize[2] << "\n" << prop.maxSurface3D[0];
		uint64_t size = frames * height * width;
		uint8_t* dev_Input,* dev_Output;

		hipMalloc((void**)&dev_Input,  size * sizeof(uint8_t));
		hipMalloc((void**)&dev_Output, size * sizeof(uint8_t));

		hipMemcpy(dev_Input, input, size * sizeof(uint8_t), hipMemcpyHostToDevice);

		uint16_t iradius = (uint16_t)radius + 0.999;

		//dim3 numBlocks(64, 8, 8);
		//dim3 threadsPerBlock(8, 8, 8);
		//SingleSDA<<<numBlocks, threadsPerBlock>>>(dev_Input, dev_Output, frames, height, width, radius, iradius, threshold, size);
		
		dim3 numBlocks(size / 1024 + 1, 1, 1);
		dim3 threadsPerBlock(1024, 1, 1);
		gpuSDAsingleDim<<<numBlocks, threadsPerBlock>>>(dev_Input, dev_Output, frames, height, width, radius, iradius, threshold, size);

		hipDeviceSynchronize();

		hipMemcpy(output, dev_Output, size * sizeof(uint8_t), hipMemcpyDeviceToHost);

		hipFree(dev_Input);
		hipFree(dev_Output);
	}

	__global__ void addKernel(int* c, const int* a, const int* b, int size) 
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < size) {
			c[i] = a[i] + b[i];
		}
	}

	void addWithCuda(int* c, const int* a, const int* b, int size) 
	{
		int* dev_a = nullptr;
		int* dev_b = nullptr;
		int* dev_c = nullptr;

		hipMalloc((void**)&dev_c, size * sizeof(int));
		hipMalloc((void**)&dev_a, size * sizeof(int));
		hipMalloc((void**)&dev_b, size * sizeof(int));

		gpuErrchk(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice));

		addKernel <<<2, (size + 1) / 2 >>> (dev_c, dev_a, dev_b, size);

		hipDeviceSynchronize();

		hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
	}
		
}