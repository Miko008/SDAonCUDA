#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>

#include ""
#include "main.h"
#include "hip/hip_runtime.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace GPU
{
	__global__ void SDAKernel3D(uint8_t* in, uint8_t* out, uint32_t frames, uint32_t height, uint32_t width, float radius, uint16_t iradius, int threshold, uint64_t size)
	{
		uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
		uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
		uint32_t z = threadIdx.z + blockIdx.z * blockDim.z;

		if (x > width || y > height || z > frames)
			return;

		for (int16_t k = -iradius; k <= iradius; k++)
			if (0 <= z + k && z + k < frames)
				for (int16_t j = -iradius; j <= iradius; j++)
					if (0 <= y + j && y + j < height)
						for (int16_t i = -iradius; i <= iradius; i++)
							if (i * i + j * j + k * k <= radius * radius && 0 <= x + i && x + i < width)
								if (in[((z + k) * height + y + j) * width + x + i] >= in[(z * height + y) * width + x] + threshold)
									out[(z * height + y) * width + x]++;
	}
	__global__ void SDAKernel1D(uint8_t* in, uint8_t* out, uint32_t frames, uint32_t height, uint32_t width, float radius, uint16_t iradius, int threshold, uint64_t size)
	{
		//todo omit using division operations
		uint64_t tempid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tempid >= size)
			return;
 		uint32_t x = (tempid) % width;
		tempid /= width;
		uint32_t y = tempid % height;
		tempid /= height;
		uint32_t z = tempid % frames;

		for (int16_t k = -iradius; k <= iradius; k++)
			if (0 <= z + k && z + k < frames)
				for (int16_t j = -iradius; j <= iradius; j++)
					if (0 <= y + j && y + j < height)
						for (int16_t i = -iradius; i <= iradius; i++)
							if (i * i + j * j + k * k <= radius * radius && 0 <= x + i && x + i < width)
								if (in[((z + k) * height + y + j) * width + x + i] >= in[(z * height + y) * width + x] + threshold)
									out[(z * height + y) * width + x]++;
	}

	//Todo fix arg templates - linker error
	//template<class InBitDepth, class OutBitDepth>
	void SDA(Image<uint8_t> input, Image<uint8_t> output, float radius, int threshold)
	{
		uint8_t* devInput,* devOutput;
		uint64_t size = input.GetSize();
		hipMalloc((void**)&devInput,  size * sizeof(uint8_t));
		hipMalloc((void**)&devOutput, size * sizeof(uint8_t));

		hipMemcpy(devInput, input.GetDataPtr(), size * sizeof(uint8_t), hipMemcpyHostToDevice);

		uint16_t iradius = std::ceil(radius);

		//dim3 numBlocks(64, 8, 8);
		//dim3 threadsPerBlock(8, 8, 8);
		//SDAKernel3D<<<numBlocks, threadsPerBlock>>>(devInput, devOutput, frames, height, width, radius, iradius, threshold, size);
		
		dim3 numBlocks(size / 1024 + 1, 1, 1);
		dim3 threadsPerBlock(1024, 1, 1);
		SDAKernel1D<<<numBlocks, threadsPerBlock>>>
			(devInput, devOutput, input.Frames(), input.Height(), input.Width(), radius, iradius, threshold, size);

		hipDeviceSynchronize();

		hipMemcpy(output.GetDataPtr(), devOutput, size * sizeof(uint8_t), hipMemcpyDeviceToHost);

		hipFree(devInput);
		hipFree(devOutput);
	}

	__device__ uint8_t CalculateDominanceOverMoreIntense(uint8_t pixel, uint16_t* histogram, uint16_t diffLen, int threshold, uint16_t* result)
	{
		if (pixel + threshold <= 0)
			return 0;

		for (uint32_t i = 0; i < pixel + threshold; i++)	//add numbers of pixels that are >= pixel + threshold
			*result += histogram[i];
	}


	__device__ uint8_t CalculateDominanceOverLessIntense(uint8_t pixel, uint16_t* histogram, uint16_t diffLen, int threshold, uint16_t* result)
	{
		uint32_t start = pixel + threshold > 0 ? pixel + threshold : 0;

		for (uint32_t i = start; i < diffLen; i++)			//add numbers of pixels that are >= pixel + threshold
			*result += histogram[i];
	}


	__global__ void FHFirstHistogramKernel(uint8_t* in, uint16_t* histogram, uint32_t frames, uint32_t height, uint32_t width, float radius, uint16_t iradius, int threshold)
	{
		uint64_t tempid = threadIdx.x + blockIdx.x * blockDim.x;
		uint32_t x = (tempid) % width;

		if (tempid < iradius || tempid > width - iradius)		//skip margins
			return;

		float asqr = radius * radius;

		for (int16_t k = -iradius; k <= iradius; k++)
			for (int16_t j = -iradius; j <= iradius; j++)
				for (int16_t i = -iradius; i <= iradius; i++)
					if (i * i + j * j + k * k <= asqr)
						histogram[x * width +												//number of histogram
						in[((iradius + k) * height + iradius + j) * width + x + i]]++;		//value of intensity to histogram 
	}


	__global__ void FHKernel(uint8_t* in, uint8_t* out, uint16_t* histogram, Coords* DiffRemZ, Coords* DiffAddZ, Coords* DiffRemY, Coords* DiffAddY, uint16_t diffLen, uint32_t frames, uint32_t height, uint32_t width, uint16_t iradius, int threshold, bool moreIntense)
	{
		//todo omit using division operations
		uint64_t tempid = threadIdx.x + blockIdx.x * blockDim.x;
		uint32_t x = (tempid) % width;

		if (x < iradius || tempid > width)		//skip margins
			return;
		//tempid /= width;\
		uint32_t y = tempid % height;\
		tempid /= height;\
		uint32_t z = tempid % frames;

		auto CalculateDominance = moreIntense ? CalculateDominanceOverLessIntense :
			CalculateDominanceOverMoreIntense;
		
		//odpalic 3 fun do __device

		for (uint32_t z = iradius; z < frames - iradius; z++)
		{
			if (z != iradius)
			{
				for (uint32_t i = 0; i < diffLen; i++)      // compute by removing and adding delta pixels to histogram
				{
					histogram[x * width +
						in[z + DiffRemZ[i].z, iradius + DiffRemZ[i].y, DiffRemZ[i].x]]--;
					histogram[x * width +
						in[z + DiffAddZ[i].z, iradius + DiffAddZ[i].y, DiffAddZ[i].x]]++;
				}
			}

			for (uint32_t y = iradius; y < height - iradius; y++)
			{
				if (y != iradius)
				{
					for (uint32_t i = 0; i < diffLen; i++)      // compute by removing and adding delta pixels to histogram
					{
						histogram[x * width +
							in[z + DiffRemY[i].z, DiffRemY[i].y, DiffRemY[i].x]]--;
						histogram[x * width +
							in[z + DiffAddY[i].z, DiffAddY[i].y, DiffAddY[i].x]]++;
					}
				}

				uint16_t result = 0;
				for (uint32_t i = 0; i < in[z, y, x] + threshold; i++)	//add numbers of pixels that are >= pixel + threshold
					result += histogram[i];
				//CalculateDominance(in[(z * height + y) * width + x], histogram, diffLen, threshold, &result);
				out[(z * height + y) * width + x] = result;
			}
		}
	}

	void FlyingHistogram(Image<uint8_t> input, Image<uint8_t> output, float radius, int threshold, bool moreIntense)
	{

		uint16_t iradius = std::ceil(radius);

		uint16_t DiffLen = 0, DiffLenZ = 0;
		Coords* DiffAddZ, * DiffRemZ, * DiffAddY, * DiffRemY, * DiffAddX, * DiffRemX;   //array of coords of delta pixels

		DiffLenZ =	SetUpRadiusDifference(radius, &DiffAddZ, &DiffRemZ, true, Direction::Z); //number of delta pixels
		DiffLen  =	SetUpRadiusDifference(radius, &DiffAddY, &DiffRemY, true, Direction::Y);
		//			SetUpRadiusDifference(radius, &DiffAddX, &DiffRemX, true, Direction::X);

		//to do anisotropic
		//float asqr = radius * radius;
		//float csqr = radiusZ * radiusZ;


		dim3 numBlocks(input.Width() / 1024 + 1, 1, 1);
		dim3 threadsPerBlock(1024, 1, 1);

		//HistogramArray<uint8_t> histogramX = HistogramArray<uint8_t>();

		uint64_t size = input.GetSize();
		uint8_t* devInput, * devOutput;

		Coords* devDiffAddZ, * devDiffRemZ, * devDiffAddY, * devDiffRemY;
		gpuErrchk(hipMalloc(&devDiffAddZ, DiffLenZ * sizeof(Coords)));
		gpuErrchk(hipMalloc(&devDiffRemZ, DiffLenZ * sizeof(Coords)));
		gpuErrchk(hipMalloc(&devDiffAddY, DiffLen  * sizeof(Coords)));
		gpuErrchk(hipMalloc(&devDiffRemY, DiffLen  * sizeof(Coords)));
		gpuErrchk(hipMemcpy(devDiffAddZ, DiffAddZ, DiffLenZ * sizeof(Coords), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(devDiffRemZ, DiffRemZ, DiffLenZ * sizeof(Coords), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(devDiffAddY, DiffAddY, DiffLen * sizeof(Coords), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(devDiffRemY, DiffRemY, DiffLen * sizeof(Coords), hipMemcpyHostToDevice));

		uint16_t* devHistogram;					//array of starter histograms - only for device memory, no need to copy to host
		//uint16_t* devHistogramCopy;				//working copy of histogram array
		uint64_t histogtamSize = sizeof(uint8_t) * input.Width() * sizeof(uint16_t);

		hipMalloc(&devHistogram, histogtamSize);
		//hipMalloc(&devHistogramCopy, histogtamSize);
		hipMalloc(&devInput, size * sizeof(uint8_t));
		hipMalloc(&devOutput, size * sizeof(uint8_t));

		gpuErrchk(hipMemcpy(devInput, input.GetDataPtr(), size * sizeof(uint8_t), hipMemcpyHostToDevice));

		uint32_t frames = input.Frames(),
				 height = input.Height(),
				 width  = input.Width();

		FHFirstHistogramKernel<<<numBlocks, threadsPerBlock>>>
			(devInput, devHistogram, frames, height, width, radius, iradius, threshold);

		//gpuErrchk(hipMemcpy(devHistogramCopy, devHistogram, histogtamSize, hipMemcpyDeviceToDevice));

		FHKernel<<<numBlocks, threadsPerBlock>>>
			(devInput, devOutput, devHistogram, devDiffRemZ, devDiffAddZ, devDiffRemY, devDiffAddY, 
				DiffLen, frames, height, width, iradius, threshold, moreIntense);

		gpuErrchk(hipMemcpy(output.GetDataPtr(), devOutput, size * sizeof(uint8_t), hipMemcpyDeviceToHost));

		hipFree(devDiffAddZ);
		hipFree(devDiffRemZ);
		hipFree(devDiffAddY);
		hipFree(devDiffRemY);

		//hipFree(devHistogramCopy);
		hipFree(devHistogram);
		hipFree(devInput);
		hipFree(devOutput);
	}

	__global__ void addKernel(int* c, const int* a, const int* b, int size) 
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < size) {
			c[i] = a[i] + b[i];
		}
	}

	void addWithCuda(int* c, const int* a, const int* b, int size) 
	{
		int* dev_a = nullptr;
		int* dev_b = nullptr;
		int* dev_c = nullptr;

		hipMalloc((void**)&dev_c, size * sizeof(int));
		hipMalloc((void**)&dev_a, size * sizeof(int));
		hipMalloc((void**)&dev_b, size * sizeof(int));

		gpuErrchk(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice));

		//addKernel<<<2, (size + 1) / 2 >>> (dev_c, dev_a, dev_b, size);

		hipDeviceSynchronize();

		gpuErrchk(hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost));

		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
	}
		
}